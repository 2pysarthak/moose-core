#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <algorithm>

#include "CudaGlobal.h"

#include "RateLookup.h"
#include "HSolveActive.h"

#ifdef USE_CUDA

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/system/system_error.h>
#include <thrust/copy.h>

__device__ __constant__ int instant_xyz_d[3];

/*
 * Copy row arrays to device.
 * To isolate CUDA functions from HSolveActive.cpp
 */
void HSolveActive::copy_to_device(double ** v_row_array, double * v_row_temp, int size)
{
	cudaSafeCall(hipMalloc((void**)v_row_array, sizeof(double) * size));
	cudaSafeCall(hipMemcpy(*v_row_array, v_row_temp, sizeof(double) * size, hipMemcpyHostToDevice));
}


/*
 * The kernel function to be executed on each CUDA thread.
 * 
 * This version uses one thread for one channel. 
 */
__global__
void advanceChannel_kernel(
	double                          * vTable,
	const unsigned                  v_nColumns,
	double							* v_row_array,
	LookupColumn                    * column_array,                      
	double                          * caTable,
	const unsigned                  ca_nColumns,
	ChannelData 					* channel,
	double                           * ca_row_array,
	double                          * istate,
	const unsigned                  channel_size,
	double                          dt,
	const unsigned					num_of_compartment
	)
{
	int tID = threadIdx.x + blockIdx.x * blockDim.x;
	int id = tID;
	if ((tID)>= channel_size) return;

	//Load channel info into thread local memory.
	u64 data = channel[tID];	

	tID = get_state_index(data);
	double myrow = v_row_array[get_compartment_index(data)];		
	double * iTable;
	unsigned inCol;
	
	bool xyz[3] = {get_x(data), get_y(data), get_z(data)};

	for(int i = 0; i < 3; ++i)
	{	
		if(!xyz[i]) continue;

		if (i == 2 && ca_row_array[get_ca_row_index(data)]!= -1.0f){
			myrow = ca_row_array[get_ca_row_index(data)];
			iTable = caTable;
			inCol = ca_nColumns;				
		}
		else {
			iTable = vTable;
			inCol = v_nColumns;
		}
		
		double a,b,C1,C2;
		double *ap, *bp;

		ap = iTable + int(myrow) + column_array[tID].column;
		bp = ap + inCol;	
		a = *ap;	
		b = *bp;
		
		C1 = a + ( b - a ) * (myrow - int(myrow));
		
		a = *( ap + 1 );		
		b = *( bp + 1 );	

		C2 = a + ( b - a ) * (myrow - int(myrow));

		/*
		 *instant_xyz_d is a CudaSymbol defined in copy_data.
		 *This array is kept in device memory as a global 
		 *constant array that can be accessed from all kernels.
		 */
		if(get_instant(data) & instant_xyz_d[i]) {
			istate[tID + i] = C1 / C2;		
		}
		
		else{
			double temp = 1.0 + dt / 2.0 * C2;
			istate[tID] = ( istate[tID] * ( 2.0 - temp ) + dt * C1 ) / temp;			
		} 
		tID ++;
	} 
}


/*
 * Copy static data from host to device,
 */
void HSolveActive::copy_data(std::vector<LookupColumn>& column,
							 LookupColumn ** 			column_dd,
							 int * 						is_inited,
							 vector<ChannelData>&		channel_data,
							 ChannelData ** 			channel_data_dd,
							 const int 					x,
							 const int 					y,
							 const int 					z)
{
	//Check if copied already.
	if(!(*is_inited))
	{
		*is_inited = 1;
		int size = column.size();
		printf("column size is :%d.\n", size);

		cudaSafeCall(hipMalloc((void**)column_dd, size * sizeof(LookupColumn)));
		cudaSafeCall(hipMemcpy(*column_dd,
								&(column.front()),
								size * sizeof(LookupColumn),
								hipMemcpyHostToDevice));
		cudaSafeCall(hipMalloc((void**)channel_data_dd, channel_data.size() * sizeof(ChannelData)));
		cudaSafeCall(hipMemcpy(*channel_data_dd,
								&(channel_data.front()),
								channel_data.size() * sizeof(ChannelData),
								hipMemcpyHostToDevice));
		const int xyz[3] = {x,y,z};
		cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(instant_xyz_d), xyz, sizeof(int)*3, 0, hipMemcpyHostToDevice));
	}	
}

/*
 * Driver function for advanceChannel calculation kernels.
 */
void HSolveActive::advanceChannel_gpu(
	double *						 v_row_d,
	vector<double>&               	 caRow,
	LookupColumn 					* column,                                           
	LookupTable&                     vTable,
	LookupTable&                     caTable,                       
	double                          * istate,
	ChannelData 					* channel,
	double                          dt,
	int 							set_size,
	int 							channel_size,
	int 							num_of_compartment
	)
{
	double * caRow_array_d;
	double * istate_d;

	int caSize = caRow.size();
	
	hipEvent_t mem_start, mem_stop;
	float mem_elapsed;
	hipEventCreate(&mem_start);
	hipEventCreate(&mem_stop);

	hipEventRecord(mem_start);

	cudaSafeCall(hipMalloc((void **)&caRow_array_d, 		caRow.size() * sizeof(double)));  
	cudaSafeCall(hipMalloc((void **)&istate_d, 			set_size * sizeof(double)));   

	cudaSafeCall(hipMemcpy(caRow_array_d, &caRow.front(), sizeof(double) * caRow.size(), hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(istate_d, istate, set_size*sizeof(double), hipMemcpyHostToDevice));
	

	//Copy static info of vTable.
	//Will only be executed once.
	if(!vTable.is_set())
	{
		vTable.set_is_set(true);
		vTable.copy_table();
	}

	//Copy static info of caTable.
	//Will only be executed once.
	if(!caTable.is_set())
	{
		caTable.set_is_set(true);
		caTable.copy_table();	
	}

	cudaCheckError();
	hipEventRecord(mem_stop);
	hipEventSynchronize(mem_stop);
	hipEventElapsedTime(&mem_elapsed, mem_start, mem_stop);

	//printf("GPU memory transfer time: %fms.\n", mem_elapsed);

	//Set kernel launch parameters.
	//BLOCK_WIDTH can be set in CudaGlobals.h
	dim3 gridSize(channel_size/BLOCK_WIDTH + 1, 1, 1);
	dim3 blockSize(BLOCK_WIDTH,1,1); 

	if(channel_size <= BLOCK_WIDTH)
	{
		gridSize.x = 1;
		blockSize.x = channel_size; 
	}    

	//Launch CUDA kernel.
	advanceChannel_kernel<<<gridSize,blockSize>>>( 
		vTable.get_table_d(),
		vTable.get_num_of_columns(),
		v_row_d,
		column,
		caTable.get_table_d(),
		caTable.get_num_of_columns(),
		channel,
		caRow_array_d,
		istate_d,
		channel_size,
		dt,
		num_of_compartment
	);

	cudaCheckError(); 

	//Copy the result from device memory back to host.
	cudaSafeCall(hipMemcpy(istate, istate_d, set_size * sizeof(double), hipMemcpyDeviceToHost));

	cudaSafeCall(hipDeviceSynchronize());    
 
 	cudaSafeCall(hipFree(v_row_d));
	cudaSafeCall(hipFree(caRow_array_d));
	cudaSafeCall(hipFree(istate_d));
}
#endif
